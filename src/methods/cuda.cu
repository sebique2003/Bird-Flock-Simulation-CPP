#include "hip/hip_runtime.h"
﻿#include "Header.h"
#include <hip/hip_runtime.h>
#include <>
#include <cmath>

// kernel pentru a calcula mișcarea boizilor
__global__ void kernel (Boid* boizi, Boid* boizi_noi, int numar_boizi) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numar_boizi) return;

	// def parametrii pentru reguli
    float aliniere_x = 0, aliniere_y = 0;
    float coeziune_x = 0, coeziune_y = 0;
    float separare_x = 0, separare_y = 0;
    int numar_aliniere = 0, numar_coeziune = 0, numar_separare = 0;

	// parcurgem boizii pentru a aplica regulile de aliniere, coeziune si separare
    for (int j = 0; j < numar_boizi; ++j) {
        if (i == j) continue;

        float dx = boizi[j].pozitie_x - boizi[i].pozitie_x;
        float dy = boizi[j].pozitie_y - boizi[i].pozitie_y;
        float distanta_patrat = dx * dx + dy * dy;

		// reguli de aliniere, coeziune si separare
        if (distanta_patrat < RAZA_ALINIERE * RAZA_ALINIERE) {
            aliniere_x += boizi[j].viteza_x;
            aliniere_y += boizi[j].viteza_y;
            numar_aliniere++;
        }
        if (distanta_patrat < RAZA_COEZUNE * RAZA_COEZUNE) {
            coeziune_x += boizi[j].pozitie_x;
            coeziune_y += boizi[j].pozitie_y;
            numar_coeziune++;
        }
        if (distanta_patrat < RAZA_SEPARARE * RAZA_SEPARARE) {
            separare_x -= dx;
            separare_y -= dy;
            numar_separare++;
        }
    }

	// actualizăm viteza și poziția boidului
    Boid boid_nou = boizi[i];

    // aliniere
    if (numar_aliniere > 0) {
        aliniere_x /= numar_aliniere;
        aliniere_y /= numar_aliniere;
        float lungime = sqrtf(aliniere_x * aliniere_x + aliniere_y * aliniere_y);
        if (lungime > 0) {
            aliniere_x = aliniere_x / lungime * VITEZA_MAXIMA;
            aliniere_y = aliniere_y / lungime * VITEZA_MAXIMA;
        }
        boid_nou.viteza_x += aliniere_x * GREUTATE_ALINIERE;
        boid_nou.viteza_y += aliniere_y * GREUTATE_ALINIERE;
    }

    // coeziune
    if (numar_coeziune > 0) {
        coeziune_x /= numar_coeziune;
        coeziune_y /= numar_coeziune;
        float dx = coeziune_x - boizi[i].pozitie_x;
        float dy = coeziune_y - boizi[i].pozitie_y;
        float lungime = sqrtf(dx * dx + dy * dy);
        if (lungime > 0) {
            dx = dx / lungime * VITEZA_MAXIMA;
            dy = dy / lungime * VITEZA_MAXIMA;
        }
        boid_nou.viteza_x += dx * GREUTATE_COEZUNE;
        boid_nou.viteza_y += dy * GREUTATE_COEZUNE;
    }

    // separare
    if (numar_separare > 0) {
        separare_x /= numar_separare;
        separare_y /= numar_separare;
        float lungime = sqrtf(separare_x * separare_x + separare_y * separare_y);
        if (lungime > 0) {
            separare_x = separare_x / lungime * VITEZA_MAXIMA;
            separare_y = separare_y / lungime * VITEZA_MAXIMA;
        }
        boid_nou.viteza_x += separare_x * GREUTATE_SEPARARE;
        boid_nou.viteza_y += separare_y * GREUTATE_SEPARARE;
    }

    // limitam viteza
    float viteza = sqrtf(boid_nou.viteza_x * boid_nou.viteza_x + boid_nou.viteza_y * boid_nou.viteza_y);
    if (viteza > VITEZA_MAXIMA) {
        boid_nou.viteza_x = (boid_nou.viteza_x / viteza) * VITEZA_MAXIMA;
        boid_nou.viteza_y = (boid_nou.viteza_y / viteza) * VITEZA_MAXIMA;
    }

    // actualizam pozitia
    boid_nou.pozitie_x += boid_nou.viteza_x;
    boid_nou.pozitie_y += boid_nou.viteza_y;

    boizi_noi[i] = boid_nou;
}

void cuda(vector<Boid>& boizi, int nr_iteratii) {
    Boid* d_boizi, * d_boizi_noi;
    int numar_boizi = static_cast<int>(boizi.size());
    size_t size = numar_boizi * sizeof(Boid);

    // alocam mem pe GPU
    hipMalloc(&d_boizi, size);
    hipMalloc(&d_boizi_noi, size);

	// copiem datele de pe CPU pe GPU
    hipMemcpy(d_boizi, boizi.data(), size, hipMemcpyHostToDevice);

	// def dimensiuni grid si bloc
    int blockSize = 256;
    int gridSize = (numar_boizi + blockSize - 1) / blockSize;

	// rulam kernel-ul pentru nr de ii specificat
    for (int i = 0; i < nr_iteratii; ++i) {
        kernel << <gridSize, blockSize >> > (d_boizi, d_boizi_noi, numar_boizi);
        hipDeviceSynchronize();

		// coperim rezultatele de la GPU inapoi pe GPU
        hipMemcpy(d_boizi, d_boizi_noi, size, hipMemcpyDeviceToDevice);
    }

    hipMemcpy(boizi.data(), d_boizi, size, hipMemcpyDeviceToHost);

	// eliberam memoria GPU
    hipFree(d_boizi);
    hipFree(d_boizi_noi);
}